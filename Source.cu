#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <openssl/md5.h>
#include <time.h>
#include <string.h>
#include <stdarg.h>

#define max_passwords 100500

void md5_hash_string(unsigned char* output, char* input, int input_len)

{

    MD5_CTX md5_ctx;

    MD5_Init(&md5_ctx);

    MD5_Update(&md5_ctx, input, input_len);

    MD5_Final(output, &md5_ctx);

}

/*
__device__ void md5_hash_string(unsigned char* output, char* input, int input_len, int (*MD5_Init)(MD5_CTX* c), int (*MD5_Update)(MD5_CTX* c, const void* data, size_t len), int (*MD5_Final)(unsigned char* md, MD5_CTX* c))
{
    MD5_CTX md5_ctx;
    MD5_Init(&md5_ctx);
    MD5_Update(&md5_ctx, input, input_len);
    MD5_Final(output, &md5_ctx);
}

*/

__device__ int strcmp_device(const char* str1, const char* str2)
{
    for (; *str1 == *str2; str1++, str2++)
        if (*str1 == '\0') 
            return 0;
    return (*(unsigned char*)str1 < *(unsigned char*)str2) ? -1 : 1;
}


__device__ size_t strcspn_device(const char* str1, const char* str2)
{
    size_t count = 0;
    while (*str1)
    {
        const char* s = str2;
        while (*s)
        {
            if (*str1 == *s)
                return count;
            s++;
        }
        str1++;
        count++;
    }
    return count;
}

__device__ int found;
__device__ char* hashed_password = "845fe25803ec3078dc795ec7187e3fd2";
//sagar at 99999
//sagara at 19275

__global__ void crackPassword(char** passwords, int (*MD5_Final)(unsigned char* md, MD5_CTX* c), int (*MD5_Init)(MD5_CTX* c), int (*MD5_Update)(MD5_CTX* c, const void* data, size_t len))
{
    int max_to_check = max_passwords;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // Remove trailing newline character

    if (i < max_passwords && found == 0 && i < max_to_check)
    {
        passwords[i][strcspn_device(passwords[i], "\n")] = 0;


        if (strcmp_device(passwords[i], hashed_password) == 0)
        {
            printf("Match found: %s at Index %d\n", passwords[i], i);
            found = i;
            max_to_check = i;
      //      max_passwords = i;

        }
    }
}

int main(int argc, char* argv[])
{
    // double start_time = omp_get_wtime();

    clock_t start = clock();

    FILE* fptr;
    // fptr = fopen("/content/drive/MyDrive/parallel2/rockyou.txt", "r");
    fptr = fopen("D:/Desktop/rockyou.txt", "r");

    if (fptr == NULL)
    {
        printf("%s", "Error reading file\n");
        return 1;
    }


    char** original_passwords = (char**)malloc(max_passwords * sizeof(char*));
    for (int i = 0; i < max_passwords; i++)
    {
        original_passwords[i] = (char*)malloc(100 * sizeof(char));
    }



    char password[100];

    char** passwords = (char**)malloc(max_passwords * sizeof(char*));

    for (int i = 0; i < max_passwords; i++)
    {
        passwords[i] = (char*)malloc(100 * sizeof(char));
    }

    char** device_passwords;

    int size = max_passwords * sizeof(char*);

    hipMalloc((void**)&device_passwords, size);

    // float *h_temp[max_passwords];
    char** h_temp[max_passwords];

    int i = 0;

    // Store the content of the file
    // Read the content
    while (fgets(password, 100, fptr) && i < max_passwords)
    {
        password[strcspn(password, "\n")] = 0;
        strcpy(original_passwords[i], password);

        // break;
        int str_len = strlen(password);

        unsigned char md5_hash[MD5_DIGEST_LENGTH];

        md5_hash_string(md5_hash, password, str_len);

        // Initialize hash and md5_hash_password arrays
        char hash[MD5_DIGEST_LENGTH * 2 + 1] = { 0 };
        char md5_hash_password[3] = { 0 };

        // Convert hash to string format
        for (int i = 0; i < MD5_DIGEST_LENGTH; i++)
        {
            sprintf(md5_hash_password, "%02x", md5_hash[i]);
            strcat(hash, md5_hash_password);
        }
        strcpy(passwords[i], hash);

        i++;
    }


    for (int i = 0; i < max_passwords; i++)
    {
        hipMalloc((void**)&h_temp[i], 100 * sizeof(char));
    }

    hipMemcpy(device_passwords, h_temp, size, hipMemcpyHostToDevice);

    for (int i = 0; i < max_passwords; i++)

    {

        hipMemcpy(h_temp[i], passwords[i], 100 * sizeof(char), hipMemcpyHostToDevice);
    }


    // crackPassword<<(max_passwords-1)/256 + 1, 256>>(device_passwords);
    crackPassword << <(max_passwords - 1) / 256 + 1, 128 >> > (device_passwords, MD5_Final, MD5_Init, MD5_Update);
    hipError_t error;
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }



    // Close the file
    fclose(fptr);

    int found_index = -1;
    hipMemcpyFromSymbol(&found_index, HIP_SYMBOL(found), sizeof(int), 0, hipMemcpyDeviceToHost);
    if (found_index != -1)
    {
        printf("Original password: %s at Index %d\n", original_passwords[found_index], found_index);
    }

    hipFree(device_passwords);
    hipFree(h_temp);
    // free allocated memory
    for (int i = 0; i < max_passwords; i++)
    {
        free(passwords[i]);
    }
    free(passwords);

    clock_t end = clock();
    double time = (double)(end - start) / CLOCKS_PER_SEC;

    printf("Cuda C Brute Force Execution time is: %.3lf sec\n",
        time);

    return 0;
}